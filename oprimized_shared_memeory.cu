#include "hip/hip_runtime.h"
//EEC 289Q final project
//Team information: Huian Wang, Minhui Huang
//Solve Car rental problem in "Reinforcement Learning: An Introduction" using dynamic programming on GPU

#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//Global Parameter Initialization

//maximum # of cars in each location
#define MAX_CARS 20
//maximum # of cars to move during night
#define MAX_MOVE_OF_CARS 5
//expectation for rental requests in first location
#define RENTAL_REQUEST_FIRST_LOC 3
//expectation for rental requests in second location
#define RENTAL_REQUEST_SECOND_LOC 4
//expectation for # of cars returned in first location
#define RETURNS_FIRST_LOC 3
//expectation for # of cars returned in second location
#define RETURNS_SECOND_LOC 2
//discount
#define DISCOUNT 0.9
//credit earned by a car
#define RENTAL_CREDIT 10
//cost of moving a car
#define MOVE_CAR_COST 2
// An up bound for poisson distribution
//If n is greater than this value, then the probability of getting n is truncated to 0
#define POISSON_UP_BOUND 11

//compute factorial
long factorial(int n) {
	if (n <= 0) {
		return 1;
	}
	else {
		return n * factorial(n - 1);
	}
}

//print matrix
void print_matrix(float *matrix, int n, int m) {
	int i, j;

	for (i = 0; i < n; i++) {
		for (j = 0; j < m; j++) {
			printf("%f   ", matrix[i*m + j]);
		}
		printf("\n");
	}
	printf("\n");
}

void print_matrix_int(int *matrix, int n, int m) {
	int i, j;

	for (i = 0; i < n; i++) {
		for (j = 0; j < m; j++) {
			printf("%d   ", matrix[i*m + j]);
		}
		printf("\n");
	}
	printf("\n");
}


__global__
void Policy_Calculation_Kernel(float *returns, int *policy, float *stateValue, float *poisson) {

	//blockIdx.x = first_place, blockIdx.y = second_place
	//threadIdx.x = rentalRequestFirstLoc, threadIdx.y = rentalRequestSeecondLoc

	int numOfCarsFirstLoc, numOfCarsSecondLoc, realRentalFirstLoc, realRentalSecondLoc, numOfCarsFirstLoc_, numOfCarsSecondLoc_;
	float reward, prob, prob_;
	int rentalRequestFirstLoc, rentalRequestSecondLoc, returnedCarsFirstLoc, returnedCarsSecondLoc;

	__shared__ float s_stateValue[(MAX_CARS + 1)*(MAX_CARS + 1)];

	for (int i=threadIdx.x; i<=MAX_CARS; i=i+POISSON_UP_BOUND) {
        for (int j=threadIdx.y; j<=MAX_CARS; j=j+POISSON_UP_BOUND) {
            s_stateValue[i*(MAX_CARS + 1) + j] = stateValue[i*(MAX_CARS + 1) + j];
        }
    }
    __syncthreads();

    __shared__ float s_poisson[4 * POISSON_UP_BOUND];

    int a = threadIdx.x;
    if (a<=3) {
        s_poisson[threadIdx.x*POISSON_UP_BOUND + threadIdx.y] = poisson[threadIdx.x*POISSON_UP_BOUND + threadIdx.y];
    }
    __syncthreads();

    __shared__ float s_returns[POISSON_UP_BOUND*POISSON_UP_BOUND];
    
    s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y]= 0;

	//cost for moving cars
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] = s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] - MOVE_CAR_COST * abs(policy[blockIdx.x*(MAX_CARS + 1) + blockIdx.y]);
		//printf("MOVE_CAR_COST %f \n", returns[blockIdx.x*(MAX_CARS + 1) + blockIdx.y]);
	}

	//printf("%f \n", returns[blockIdx.x*(MAX_CARS + 1) + blockIdx.y]);
	numOfCarsFirstLoc = min(blockIdx.x - policy[blockIdx.x*(MAX_CARS + 1) + blockIdx.y], MAX_CARS);
	numOfCarsSecondLoc = min(blockIdx.y + policy[blockIdx.x*(MAX_CARS + 1) + blockIdx.y], MAX_CARS);
	//printf("%d \n", numOfCarsFirstLoc);
	rentalRequestFirstLoc = threadIdx.x;
	rentalRequestSecondLoc = threadIdx.y;

	// valid rental requests should be less than actual # of cars
	realRentalFirstLoc = min(numOfCarsFirstLoc, rentalRequestFirstLoc);
	realRentalSecondLoc = min(numOfCarsSecondLoc, rentalRequestSecondLoc);

	// get credits for renting
	reward = (realRentalFirstLoc + realRentalSecondLoc) * RENTAL_CREDIT;
	numOfCarsFirstLoc = numOfCarsFirstLoc - realRentalFirstLoc;
	numOfCarsSecondLoc = numOfCarsSecondLoc - realRentalSecondLoc;

	// probability for current combination of rental requests
	prob = s_poisson[0 * POISSON_UP_BOUND + rentalRequestFirstLoc] * s_poisson[1 * POISSON_UP_BOUND + rentalRequestSecondLoc];

	//record # of cars in each location and prob
	numOfCarsFirstLoc_ = numOfCarsFirstLoc;
	numOfCarsSecondLoc_ = numOfCarsSecondLoc;
	prob_ = prob;

	//consider the returned cars case
	for (returnedCarsFirstLoc = 0; returnedCarsFirstLoc < POISSON_UP_BOUND; returnedCarsFirstLoc++) {
		for (returnedCarsSecondLoc = 0; returnedCarsSecondLoc < POISSON_UP_BOUND; returnedCarsSecondLoc++) {
			numOfCarsFirstLoc = numOfCarsFirstLoc_;
			numOfCarsSecondLoc = numOfCarsSecondLoc_;
			prob = prob_;
			numOfCarsFirstLoc = min(numOfCarsFirstLoc + returnedCarsFirstLoc, MAX_CARS);
			numOfCarsSecondLoc = min(numOfCarsSecondLoc + returnedCarsSecondLoc, MAX_CARS);
			prob = prob * s_poisson[2 * POISSON_UP_BOUND + returnedCarsFirstLoc] * s_poisson[3 * POISSON_UP_BOUND + returnedCarsSecondLoc];
			
			s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] = s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] + prob * (reward + DISCOUNT * s_stateValue[numOfCarsFirstLoc*(MAX_CARS + 1) + numOfCarsSecondLoc]);
		}
	}
	returns[(blockIdx.x*POISSON_UP_BOUND+threadIdx.x)*(MAX_CARS + 1)*POISSON_UP_BOUND + blockIdx.y*POISSON_UP_BOUND+threadIdx.y] = s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y];
	//printf("%d %d %d %d %f \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, returns[(blockIdx.x*POISSON_UP_BOUND+threadIdx.x)*(MAX_CARS + 1)*POISSON_UP_BOUND + blockIdx.y*POISSON_UP_BOUND+threadIdx.y]);
}



__global__
void Policy_Calculation_Kernel2(float *returns, int action, float *stateValue, float *poisson) {

	//blockIdx.x = first_place, blockIdx.y = second_place
	//threadIdx.x = rentalRequestFirstLoc, threadIdx.y = rentalRequestSeecondLoc

	int numOfCarsFirstLoc, numOfCarsSecondLoc, realRentalFirstLoc, realRentalSecondLoc, numOfCarsFirstLoc_, numOfCarsSecondLoc_;
	float reward, prob, prob_;
	int rentalRequestFirstLoc, rentalRequestSecondLoc, returnedCarsFirstLoc, returnedCarsSecondLoc;

	__shared__ float s_stateValue[(MAX_CARS + 1)*(MAX_CARS + 1)];

	for (int i=threadIdx.x; i<=MAX_CARS; i=i+POISSON_UP_BOUND) {
        for (int j=threadIdx.y; j<=MAX_CARS; j=j+POISSON_UP_BOUND) {
            s_stateValue[i*(MAX_CARS + 1) + j] = stateValue[i*(MAX_CARS + 1) + j];
        }
    }
    __syncthreads();

    __shared__ float s_poisson[4 * POISSON_UP_BOUND];
    
    int c = threadIdx.x;
    if (c<=3) {
        s_poisson[threadIdx.x*POISSON_UP_BOUND + threadIdx.y] = poisson[threadIdx.x*POISSON_UP_BOUND + threadIdx.y];
    }
    __syncthreads();

    __shared__ float s_returns[POISSON_UP_BOUND*POISSON_UP_BOUND];
    
    s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y]= 0;

	//printf("%d \n", action);
	int a=blockIdx.x, b=blockIdx.y;
	if (action <= a && (-action) <= b) {
		//printf("%d \n", action);
		//cost for moving cars
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] = s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] - MOVE_CAR_COST * abs(action);
		}

		numOfCarsFirstLoc = min(blockIdx.x - action, MAX_CARS);
		numOfCarsSecondLoc = min(blockIdx.y + action, MAX_CARS);

		rentalRequestFirstLoc = threadIdx.x;
		rentalRequestSecondLoc = threadIdx.y;
		//printf("%d \n", action);
		// valid rental requests should be less than actual # of cars
		realRentalFirstLoc = min(numOfCarsFirstLoc, rentalRequestFirstLoc);
		realRentalSecondLoc = min(numOfCarsSecondLoc, rentalRequestSecondLoc);

		// get credits for renting
		reward = (realRentalFirstLoc + realRentalSecondLoc) * RENTAL_CREDIT;
		numOfCarsFirstLoc = numOfCarsFirstLoc - realRentalFirstLoc;
		numOfCarsSecondLoc = numOfCarsSecondLoc - realRentalSecondLoc;

		// probability for current combination of rental requests
		prob = s_poisson[0 * POISSON_UP_BOUND + rentalRequestFirstLoc] * s_poisson[1 * POISSON_UP_BOUND + rentalRequestSecondLoc];

		//record # of cars in each location and prob
		numOfCarsFirstLoc_ = numOfCarsFirstLoc;
		numOfCarsSecondLoc_ = numOfCarsSecondLoc;
		prob_ = prob;

		//consider the returned cars case
		for (returnedCarsFirstLoc = 0; returnedCarsFirstLoc < POISSON_UP_BOUND; returnedCarsFirstLoc++) {
			for (returnedCarsSecondLoc = 0; returnedCarsSecondLoc < POISSON_UP_BOUND; returnedCarsSecondLoc++) {
				numOfCarsFirstLoc = numOfCarsFirstLoc_;
				numOfCarsSecondLoc = numOfCarsSecondLoc_;
				prob = prob_;
				numOfCarsFirstLoc = min(numOfCarsFirstLoc + returnedCarsFirstLoc, MAX_CARS);
				numOfCarsSecondLoc = min(numOfCarsSecondLoc + returnedCarsSecondLoc, MAX_CARS);
				prob = prob * s_poisson[2 * POISSON_UP_BOUND + returnedCarsFirstLoc] * s_poisson[3 * POISSON_UP_BOUND + returnedCarsSecondLoc];
				s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] = s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y] + prob * (reward + DISCOUNT * s_stateValue[numOfCarsFirstLoc*(MAX_CARS + 1) + numOfCarsSecondLoc]);
			}
		}
		//printf("%d \n", action);
		//printf("INSIDE: %d %d %d %f \n", action, blockIdx.x, blockIdx.y, returns[(blockIdx.x*POISSON_UP_BOUND+threadIdx.x)*(MAX_CARS + 1)*POISSON_UP_BOUND + blockIdx.y*POISSON_UP_BOUND+threadIdx.y]);
	}
	returns[(blockIdx.x*POISSON_UP_BOUND+threadIdx.x)*(MAX_CARS + 1)*POISSON_UP_BOUND + blockIdx.y*POISSON_UP_BOUND+threadIdx.y] = s_returns[threadIdx.x*POISSON_UP_BOUND+threadIdx.y];
	//printf("INSIDE: %d %d %d %f \n", action, blockIdx.x, blockIdx.y, returns[(blockIdx.x*POISSON_UP_BOUND+threadIdx.x)*(MAX_CARS + 1)*POISSON_UP_BOUND + blockIdx.y*POISSON_UP_BOUND+threadIdx.y]);
}



int main() {
	float *returns;

	int *policy;

	float *stateValue;

	float *poisson;

	int old_action[(MAX_CARS + 1)*(MAX_CARS + 1)];

	float temp[(MAX_CARS + 1)*(MAX_CARS + 1)];

	int i, j, k, n, action;
	float v;

	size_t size1 = (MAX_CARS + 1)*(MAX_CARS + 1) * sizeof(float);
	size_t size2 = (MAX_CARS + 1)*(MAX_CARS + 1) * sizeof(int);
	size_t size3 = 4 * POISSON_UP_BOUND * sizeof(float);
	size_t size4 = (MAX_CARS + 1)*(MAX_CARS + 1)*  POISSON_UP_BOUND * POISSON_UP_BOUND * sizeof(float);
	hipMallocManaged(&returns, size4);
	hipMallocManaged(&policy, size2);
	hipMallocManaged(&stateValue, size1);
	hipMallocManaged(&poisson, size3);

	for (i = 0; i <= MAX_CARS; i++) {
		for (j = 0; j <= MAX_CARS; j++) {
			policy[i*(MAX_CARS + 1) + j] = 0;
			stateValue[i*(MAX_CARS + 1) + j] = 0;
			old_action[i*(MAX_CARS + 1) + j] = 0;
			temp[i*(MAX_CARS + 1) + j] = -1.0e38;
		}
	}

	for (i = 0; i < (MAX_CARS+1)*POISSON_UP_BOUND; i++) {
		for (j = 0; j < (MAX_CARS+1)*POISSON_UP_BOUND; j++) {
			returns[i*(MAX_CARS + 1)*POISSON_UP_BOUND + j] = 0;
		}
	}
	//print_matrix(stateValue, MAX_CARS + 1, MAX_CARS + 1);
	// Probability for poisson distribution

	for (i = 0; i < POISSON_UP_BOUND; i++) {
		poisson[0 * POISSON_UP_BOUND + i] = exp(-RENTAL_REQUEST_FIRST_LOC) * pow(RENTAL_REQUEST_FIRST_LOC, i) / factorial(i);
		poisson[1 * POISSON_UP_BOUND + i] = exp(-RENTAL_REQUEST_SECOND_LOC) * pow(RENTAL_REQUEST_SECOND_LOC, i) / factorial(i);
		poisson[2 * POISSON_UP_BOUND + i] = exp(-RETURNS_FIRST_LOC) * pow(RETURNS_FIRST_LOC, i) / factorial(i);
		poisson[3 * POISSON_UP_BOUND + i] = exp(-RETURNS_SECOND_LOC) * pow(RETURNS_SECOND_LOC, i) / factorial(i);
	}

	//printf("%f\n", poisson[0]);

	//policy iteration
	bool policy_stable = false;

	float tolerance = 0.0001;

	while (policy_stable == false) {

		//Policy iteration
		while (true) {

			float delta = 0;

			dim3 dimBlock(POISSON_UP_BOUND, POISSON_UP_BOUND);
			dim3 dimGrid(MAX_CARS + 1 , MAX_CARS + 1);
			Policy_Calculation_Kernel <<<dimGrid, dimBlock>>>(returns, policy, stateValue, poisson); //launch kernel

			hipDeviceSynchronize();
			
			//print_matrix(returns, (MAX_CARS + 1)*POISSON_UP_BOUND, (MAX_CARS + 1)*POISSON_UP_BOUND);
			
			for (i = 0; i <= MAX_CARS; i++) {
				for (j = 0; j <= MAX_CARS; j++) {
					//stateValue[i*(MAX_CARS + 1) + j] = returns[i*(MAX_CARS + 1) + j];
					v = 0;
					for (k = 0; k < POISSON_UP_BOUND; k++) {
						for (n = 0; n < POISSON_UP_BOUND; n++) {
							v = v + returns[(i*POISSON_UP_BOUND + k)*(MAX_CARS + 1)*POISSON_UP_BOUND + j * POISSON_UP_BOUND + n];
							returns[(i*POISSON_UP_BOUND + k)*(MAX_CARS + 1)*POISSON_UP_BOUND + j * POISSON_UP_BOUND + n] = 0;
						}
					}
					
					delta = max(delta, abs(v - stateValue[i*(MAX_CARS + 1) + j]));
					stateValue[i*(MAX_CARS + 1) + j] = v;
				}
			}

			//print_matrix(stateValue, MAX_CARS + 1, MAX_CARS + 1);
			//break;
			//printf("%f\n", delta);

			if (delta < tolerance) {
				//printf("%f\n", delta);
				//print_matrix(stateValue, MAX_CARS + 1, MAX_CARS + 1);
				break;
			}
		}
		//break;

		//Policy inprovement GPU
		policy_stable = true;

		for (action = -MAX_MOVE_OF_CARS; action <= MAX_MOVE_OF_CARS; action++) {

			dim3 dimBlock(POISSON_UP_BOUND, POISSON_UP_BOUND);
			dim3 dimGrid(MAX_CARS + 1, MAX_CARS + 1);
			Policy_Calculation_Kernel2 <<<dimGrid, dimBlock>>>(returns, action, stateValue, poisson);

			hipDeviceSynchronize();
			//printf("OUTSIDE\n");
			//print_matrix(returns, (MAX_CARS + 1)*POISSON_UP_BOUND, (MAX_CARS + 1)*POISSON_UP_BOUND);
			//printf("****************\n");

			for (i = 0; i <= MAX_CARS; i++) {
				for (j = 0; j <= MAX_CARS; j++) {
					v = 0;
					for (k = 0; k < POISSON_UP_BOUND; k++) {
						for (n = 0; n < POISSON_UP_BOUND; n++) {
							v = v + returns[(i*POISSON_UP_BOUND + k)*(MAX_CARS + 1)*POISSON_UP_BOUND + j * POISSON_UP_BOUND + n];
							returns[(i*POISSON_UP_BOUND + k)*(MAX_CARS + 1)*POISSON_UP_BOUND + j * POISSON_UP_BOUND + n] = 0;
						}
					}
					if (v > temp[i*(MAX_CARS + 1) + j]) {
						temp[i*(MAX_CARS + 1) + j] = v;
						policy[i*(MAX_CARS + 1) + j] = action;
					}
					
				}
			}

		}
		
		for (i = 0; i <= MAX_CARS; i++) {
			for (j = 0; j <= MAX_CARS; j++) {
				if (old_action[i*(MAX_CARS + 1) + j] != policy[i*(MAX_CARS + 1) + j]) {
					policy_stable = false;
					old_action[i*(MAX_CARS + 1) + j] = policy[i*(MAX_CARS + 1) + j];
				}
				temp[i*(MAX_CARS + 1) + j] = -1.0e38;
			}
		}

		print_matrix_int(policy, MAX_CARS + 1, MAX_CARS + 1);

	}

}
